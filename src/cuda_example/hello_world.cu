#include "hip/hip_runtime.h"
#include "hello_world/hello_world.h"

__global__
void hello(char *a, int *b)
{
  //a[threadIdx.x] += b[threadIdx.x];
  a[threadIdx.x] = sharedFunction(a[threadIdx.x], b[threadIdx.x]);

}

int testmainCUDA()
{
  char a[N] = "Hello ";
  int b[N] = {15, 10, 6, 0, -11, 1, 0}; 
  char *ad;
  int *bd;
  const int csize = N*sizeof(char);
  const int isize = N*sizeof(int);
  
  printf("%s", a);
  
  if ( hipSuccess != hipMalloc( (void**)&ad, csize ) )
    printf( "cannot allocate device memory to ad\n");
  if ( hipSuccess != hipMalloc( (void**)&bd, isize ) )
    printf( "cannot allocate device memory to bd\n");
  if ( hipSuccess != hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ) )
    printf( "cannot copy memory to device\n");
  if ( hipSuccess != hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ) )
    printf( "cannot copy memory to device\n");
  
  dim3 dimBlock( blocksize, 1 );
  dim3 dimGrid( 1, 1 );
  hello<<<dimGrid, dimBlock>>>(ad, bd);
  hipError_t err = hipGetLastError();
  if ( hipSuccess != err )
  {
    fprintf( stderr, "cudaCheckError() %d:%s\n", err, hipGetErrorString( err ) );
    exit( -1 );
  }
  if ( hipSuccess != hipGetLastError() )
        printf( "Kernel Error!\n" );
  hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
  hipFree( ad );
  
  printf("%s from CUDA\n", a);
  return EXIT_SUCCESS;
}
