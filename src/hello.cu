#include "hip/hip_runtime.h"
#include "hello.h"

__global__
void hello(char *a, int *b)
{
    a[threadIdx.x] += b[threadIdx.x];
}

int testmain()
{
  char a[N] = "Hello ";
  int b[N] = {15, 10, 6, 0, -11, 1, 0};
  
  char *ad;
  int *bd;
  const int csize = N*sizeof(char);
  const int isize = N*sizeof(int);
  
  printf("%s", a);
  
  if ( hipSuccess != hipMalloc( (void**)&ad, csize ) )
    printf( "cannot allocate device memory to ad");
  if ( hipSuccess != hipMalloc( (void**)&bd, isize ) )
    printf( "cannot allocate device memory to bd");
  if ( hipSuccess != hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ) )
    printf( "cannot copy memory to devide");
  if ( hipSuccess != hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ) )
    printf( "cannot copy memory to devide");
  
  dim3 dimBlock( blocksize, 1 );
  dim3 dimGrid( 1, 1 );
  hello<<<dimGrid, dimBlock>>>(ad, bd);
  hipError_t err = hipGetLastError();
  if ( hipSuccess != err )
  {
    fprintf( stderr, "cudaCheckError() %d:%s\n", err, hipGetErrorString( err ) );
    exit( -1 );
  }
  if ( hipSuccess != hipGetLastError() )
        printf( "Kernel Error!\n" );
  hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
  hipFree( ad );
  
  printf("%s\n", a);
  return EXIT_SUCCESS;
}
