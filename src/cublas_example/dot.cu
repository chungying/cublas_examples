#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

int main(void)
{
    const int nvals = 10;
    const size_t sz = sizeof(double) * (size_t)nvals;
    double x[nvals], y[nvals];
    double *x_, *y_, *result_;
    double result=0., resulth=0.;

    for(int i=0; i<nvals; i++) {
        x[i] = y[i] = (double)(i)/(double)(nvals);
        resulth += x[i] * y[i];
        printf("%f ",resulth);
    }
    printf("\n");

    hipblasHandle_t h;
    hipblasCreate(&h);
    hipblasSetPointerMode(h, HIPBLAS_POINTER_MODE_DEVICE);

    hipMalloc( (void **)(&x_), sz);
    hipMalloc( (void **)(&y_), sz);
    hipMalloc( (void **)(&result_), sizeof(double) );

    hipMemcpy(x_, x, sz, hipMemcpyHostToDevice);
    hipMemcpy(y_, y, sz, hipMemcpyHostToDevice);

    printf("%p, %p, %p\n",x_, y_, result_);
    hipblasDdot(h, nvals-2, x_+2, 1, y_, 1, result_);

    hipMemcpy(&result, result_, sizeof(double), hipMemcpyDeviceToHost);

    printf("%f %f\n", resulth, result);

    return 0;
}
