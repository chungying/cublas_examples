//Example 2. Application Using C and CUBLAS: 0-based indexing
//-----------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define M 6
#define N 5
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

static __inline__ void modify (
  hipblasHandle_t handle,//status
  float *m,//device array pointer 
  int ldm,//first dimension, 6
  int n,//second dimenstion, 5
  int p,//1 
  int q,//2
  float alpha,//scalar
  float beta)//scalar
{
  printf ("element_size: %d, scalar: %f, strid: %d\n", n-p, alpha, ldm);
  hipblasSscal (handle, n-q, &alpha, &m[IDX2C(p,q,ldm)], ldm);
  hipblasSscal (handle, ldm-p, &beta, &m[IDX2C(p,q,ldm)], 1);
}

int main (void){
    hipError_t cudaStat;    
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int i, j;
    float* devPtrA;
    float* a = 0;
    a = (float *)malloc (M * N * sizeof (*a));
    if (!a) {
        printf ("host memory allocation failed\n");
        return EXIT_FAILURE;
    }
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            //a[IDX2C(i,j,M)] = (float)(i * N + j + 1);
            a[IDX2C(i,j,M)] = (float)(IDX2C(i,j,M));
            //printf ("%7.0f", a[IDX2C(i,j,M)]);
        }
        //printf ("\n");
    }
    printf("original a\n");
    for (i = 0 ; i < N*M ; i++)
    {
      printf("%7.0f", a[i]);
      if(i%M==M-1)
        printf("\n");
    }
    cudaStat = hipMalloc ((void**)&devPtrA, M*N*sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed\n");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    modify (handle, devPtrA, M, N, 1, 2, 16.0f, 12.0f);

    stat = hipblasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed\n");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipFree (devPtrA);
    hipblasDestroy(handle);
    printf("modified a\n");
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            printf ("%7.0f", a[IDX2C(i,j,M)]);
        }
        printf ("\n");
    }
    free(a);
    return EXIT_SUCCESS;
}
