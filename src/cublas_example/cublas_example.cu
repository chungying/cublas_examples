//Example 2. Application Using C and CUBLAS: 0-based indexing
//-----------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
//#define M 6
//#define N 5
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

static __inline__ void modify (
  hipblasHandle_t handle,//status
  double *m,//device array pointer 
  int ldm,//first dimension, 6
  int n,//second dimenstion, 5
  int p,//1 
  int q,//2
  double alpha,//scalar
  double beta)//scalar
{
  printf ("element_size: %d, scalar: %f, strid: %d\n", n-p, alpha, ldm);
  hipblasDscal (handle, n-q, &alpha, m+IDX2C(p,q,ldm), ldm);
  hipblasDscal (handle, ldm-p, &beta, m+IDX2C(p,q,ldm), 1);
  //cublasDdot(handle, ldm, m+IDX2C(0,0,ldm), 1, m+IDX2C(0,1,ldm), 1, m+IDX2C(0,2,ldm));
}

int main (void){
    int M = 6;
    int N = 5;
    hipError_t cudaStat;    
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int i, j;
    double* devPtrA;
    double* a = 0;
    a = (double *)malloc (M * N * sizeof (*a));
    if (!a) {
        printf ("host memory allocation failed\n");
        return EXIT_FAILURE;
    }
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            a[IDX2C(i,j,M)] = (double)(IDX2C(i,j,M));
            //printf ("%7.0f", a[IDX2C(i,j,M)]);
        }
        //printf ("\n");
    }
    printf("original a\n");
    for (i = 0 ; i < N*M ; i++)
    {
      printf("%7.0f", a[i]);
      if(i%M==M-1)
        printf("\n");
    }
    cudaStat = hipMalloc ((void**)&devPtrA, M*N*sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed\n");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    printf ("pointers: %d, %d, %d\n",IDX2C(0,0,M),IDX2C(0,1,M),IDX2C(0,2,M));
    printf ("pointers: %p, %p, %p\n",a+IDX2C(0,0,M),a+IDX2C(0,1,M),a+IDX2C(0,2,M));
    modify (handle, devPtrA, M, N, 1, 2, 16.0f, 12.0f);

    stat = hipblasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed\n");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipFree (devPtrA);
    hipblasDestroy(handle);
    printf("modified a\n");
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            printf ("%7.0f", a[IDX2C(i,j,M)]);
        }
        printf ("\n");
    }
    free(a);
    return EXIT_SUCCESS;
}
