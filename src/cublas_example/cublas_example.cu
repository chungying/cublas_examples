//Example 2. Application Using C and CUBLAS: 0-based indexing
//-----------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define M 6
#define N 5
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

static __inline__ void modify (
  hipblasHandle_t handle,//status
  float *m,//device array pointer 
  int ldm,//first dimension, 6
  int n,//second dimenstion, 5
  int p,//1 
  int q,//2
  float alpha,//scalar
  float beta)//scalar
{
  hipblasSscal (handle, n-p, &alpha, &m[IDX2C(p,q,ldm)], ldm);
  hipblasSscal (handle, ldm-p, &beta, &m[IDX2C(p,q,ldm)], 1);
}

int main (void){
    hipError_t cudaStat;    
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int i, j;
    float* devPtrA;
    float* a = 0;
    a = (float *)malloc (M * N * sizeof (*a));
    if (!a) {
        printf ("host memory allocation failed\n");
        return EXIT_FAILURE;
    }
    //printf("original a\n");
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            a[IDX2C(i,j,M)] = (float)(i * M + j + 1);
            //printf ("%7.0f", a[IDX2C(i,j,M)]);
        }
        //printf ("\n");
    }
    cudaStat = hipMalloc ((void**)&devPtrA, M*N*sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed\n");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    modify (handle, devPtrA, M, N, 1, 2, 16.0f, 12.0f);

    stat = hipblasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed\n");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipFree (devPtrA);
    hipblasDestroy(handle);
    printf("modified a\n");
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            printf ("%7.0f", a[IDX2C(i,j,M)]);
        }
        printf ("\n");
    }
    free(a);
    return EXIT_SUCCESS;
}
